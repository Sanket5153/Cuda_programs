
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Size of array
#define N 5120*10000

// Kernel
__global__ void add_vectors(float *a, float *b, float *c)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < N) c[id] = a[id] + b[id];
}

// Main program
int main()
{
    // Number of bytes to allocate for N floats
    size_t bytes = N*sizeof(float);

    // Allocate memory for arrays A, B, and C on host
    float *A = (float*)malloc(bytes);
    float *B = (float*)malloc(bytes);
    float *C = (float*)malloc(bytes);

    // Allocate memory for arrays d_A, d_B, and d_C on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Fill host arrays A and B
    for(int i=0; i<N; i++)
    {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // Copy data from host arrays A and B to device arrays d_A and d_B
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Set execution configuration parameters
    //      thr_per_blk: number of CUDA threads per grid block
    //      blk_in_grid: number of blocks in grid
    int thr_per_blk = 128;
    int blk_in_grid = ceil( float(N) / thr_per_blk );

    // Launch kernel
    add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C);

    // Copy data from device array d_C to host array C
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    // Verify results
    float tolerance = 1.0e-5;
    for(int i=0; i<N; i++)
    {
        if( fabs(C[i] - 3.0f) > tolerance)
        {
            printf("\nError: value of C[%d] = %f instead of 3.0\n\n", i, C[i]);
            exit(1);
        }
    }

    // Free CPU memory
    free(A);
    free(B);
    free(C);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\n---------------------------\n");
    printf("__SUCCESS__\n");
    printf("---------------------------\n");
    printf("N                 = %d\n", N);
    printf("Threads Per Block = %d\n", thr_per_blk);
    printf("Blocks In Grid    = %d\n", blk_in_grid);
    printf("---------------------------\n\n");

    return 0;
}
